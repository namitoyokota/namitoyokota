#include "hip/hip_runtime.h"
#include "matrix.cuh"

void printmatrix(float *matrix, int row, int col)
{
  int i, j;
  printf("dimensions: %d x %d\n", row, col);
  for (i=0; i<row; i++) {
    for (j=0; j<col; j++) {
      printf("%f\t", matrix[j*row+i]);
    }
    printf("\n");
  }
  printf("\n");
}

float* readmatrix(char* filename, int *row, int *col)
{
  FILE* fp = fopen(filename,"r");

  fread(row, sizeof(int), 1, fp);
  fread(col, sizeof(int), 1, fp);

  float *matrix = (float *)calloc((*row)*(*col), sizeof(float));
  fread(matrix, sizeof(float), (*row)*(*col), fp);
  fclose(fp);

  return matrix;
}

void writematrix(const char* header, const char* output, float* matrix, int row, int col)
{
  char filename[100];
  strcpy(filename, header);
  strcat(filename, output);
  FILE* fp = fopen(filename, "w");

  fwrite(&row, sizeof(int), 1, fp);
  fwrite(&col, sizeof(int), 1, fp);

  fwrite(matrix, sizeof(float), row*col, fp);
  fclose(fp);
}

float cublas_multiply(const float *a, const float *b, float *c, const int m, const int k, const int n) {
  int lda = m, ldb = k, ldc = m;
  const float alf = 1;
  const float bet = 0;
  const float *alpha = &alf;
  const float *beta = &bet;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float milliseconds = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipblasDestroy(handle);

  return milliseconds;
}

__global__ void shared_multiply(float *Md, float *Nd, float * Pd, int a_row, int a_col, int b_col)
{
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x, by = blockIdx.y;
  int tx = threadIdx.x, ty = threadIdx.y;
  int Row = by*TILE_WIDTH+ty;
  int Col = bx*TILE_WIDTH+tx;
  float Pvalue = 0;

  for (int m = 0; m < (a_col-1)/TILE_WIDTH+1; ++m) {
       if (Row < a_row  && m*TILE_WIDTH+tx < a_col)
          Mds[ty][tx] = Md[Row+a_row*(m*TILE_WIDTH+tx)];
       else Mds[ty][tx] = 0;
       if (Col < b_col && m*TILE_WIDTH+ty < a_row)
          Nds[ty][tx] = Nd[(m*TILE_WIDTH+ty)+b_col*Col];
       else Nds[ty][tx] = 0;

       __syncthreads();
       for (int k = 0; k < TILE_WIDTH; ++k)
          Pvalue += Mds[ty][k] * Nds[k][tx];
       __syncthreads();
  }

  if (Row < a_row  && Col < b_col)
    Pd[((blockIdx.y * blockDim.y + threadIdx.y) + (blockIdx.x * blockDim.x + threadIdx.x)*b_col)] = Pvalue;
}

extern "C"
void wrapper(int argc, char *argv[])
{
  // check command line input
  if (argc != 4) {
    printf("invalid command line arguments :(\n");
    return;
  }

  // rows and columns
  int a_row, a_col;
  int b_row, b_col;

  // host arrays
  float *h_a = readmatrix(argv[1], &a_row, &a_col);
  float *h_b = readmatrix(argv[2], &b_row, &b_col);
  float *h_c = (float *)malloc(a_row*b_col*sizeof(float));

  // device arrays
  float *d_a;
  float *d_b;
  float *d_c;

  // allocate on GPU
  hipMalloc(&d_a, a_row*a_col*sizeof(float));
  hipMalloc(&d_b, b_row*b_col*sizeof(float));
  hipMalloc(&d_c, a_row*b_col*sizeof(float));

  // CPU to GPU
  hipMemcpy(d_a, h_a, a_row*a_col*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, b_row*b_col*sizeof(float), hipMemcpyHostToDevice);

  // print input matrices
  // printmatrix(h_a, a_row, a_col);
  // printmatrix(h_b, b_row, b_col);

  // dgemm using cublas
  float milliseconds = cublas_multiply(d_a, d_b, d_c, a_row, a_col, b_col);
  hipMemcpy(h_c, d_c, a_row*b_col*sizeof(float), hipMemcpyDeviceToHost);
  writematrix("cublas_", argv[3], h_c, a_row, b_col);
  // printmatrix(h_c, a_row, b_col);
  printf("cublas (%s) :\n\t%f seconds\n\t%.10f gflops\n\n", argv[3],  milliseconds/1000, (2*((double)(a_row*a_col*b_col)))/(milliseconds*1000000));

  // gemm using shared memory
  dim3 dimGrid((b_col-1)/TILE_WIDTH+1, (a_row-1)/TILE_WIDTH+1, 1);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  shared_multiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, a_row, a_col, b_col);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipMemcpy(h_c, d_c, a_row*b_col*sizeof(float), hipMemcpyDeviceToHost);
  writematrix("shared_", argv[3], h_c, a_row, b_col);
  // printmatrix(h_c, a_row, b_col);
  printf("shared (%s) :\n\t%f seconds\n\t%.10f gflops\n\n", argv[3],  milliseconds/1000, (2*((double)(a_row*a_col*b_col)))/(milliseconds*1000000));

  // free GPU memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // free CPU memory
  free(h_a);
  free(h_b);
  free(h_c);
}